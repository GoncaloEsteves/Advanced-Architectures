
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <mm_malloc.h>
#include <iostream>
#include <sys/time.h>

using namespace std;

#define GRIDSIZE 128
#define BLOCKSIZE 16

hipEvent_t start, stop;
timeval t;

void fillMatrices (float **m1, float **m2, float **result, int N) {
	*m1 = (float *)_mm_malloc(N*N * sizeof(float), 32);
    *m2 = (float *)_mm_malloc(N*N * sizeof(float), 32);
    *result = (float *)_mm_malloc(N*N * sizeof(float), 32);

	srand(time(NULL));

	for (unsigned i = 0; i < N*N; i++) {
		(*m1)[i] = (float) rand();
		(*m2)[i] = 1.0;
	}
}

int validate(float *m, int N) {
    for(unsigned i = 0; i < N*N; i += N) {
        float aux = m[i];
        for(unsigned j = 0; j < N; j++) {
            if(m[i + j] != aux) 
				return 0;
        }
    }
    return 1;
}

void startKernelTime (void) {
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

void stopKernelTime (void) {
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
	float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << milliseconds << " ms" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString( err) << endl;
		exit(-1);
	}
}

__global__
void matrixMultKernel (float *dev_m1, float *dev_m2, float *dev_res, int N) {
    __shared__ float tileM1[BLOCKSIZE][BLOCKSIZE];
    __shared__ float tileM2[BLOCKSIZE][BLOCKSIZE];

    int i = blockIdx.y * BLOCKSIZE + threadIdx.y;
    int j = blockIdx.x * BLOCKSIZE + threadIdx.x;
    float aux = 0;
    int idx;

    if (i < N && j < N){
		for (int kk = 0; kk < GRIDSIZE; kk++){
        	idx = i * N + (kk * BLOCKSIZE + threadIdx.x);

        	if (idx >= N * N)
            	tileM1[threadIdx.y][threadIdx.x] = 0;
        	else
            	tileM1[threadIdx.y][threadIdx.x] = dev_m1[idx];

        	idx = (kk * BLOCKSIZE + threadIdx.y) * N + j;

        	if (idx >= N * N)
            	tileM2[threadIdx.y][threadIdx.x] = 0;
        	else
            	tileM2[threadIdx.y][threadIdx.x] = dev_m2[idx];

        	__syncthreads();

        	for (int k = 0; k < BLOCKSIZE; k++)
            	aux += tileM1[threadIdx.y][k] * tileM2[k][threadIdx.x];
				
        	__syncthreads();
    	}

        dev_res[i * N + j] = aux;
    }
}

void matrixMultGPU (float *m1, float *m2, float *result, int N) {

    float *dev_m1, *dev_m2, *dev_res;

    hipMalloc((void **)&dev_m1, N*N * sizeof(float));
    hipMalloc((void **)&dev_m2, N*N * sizeof(float));
	hipMalloc((void **)&dev_res, N*N * sizeof(float));

    startKernelTime();
    hipMemcpy(dev_m1, m1, N*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_m2, m2, N*N * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("Error using hipMemcpyHostToDevice");
    stopKernelTime();

	dim3 dimGrid(GRIDSIZE, GRIDSIZE);
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);

    startKernelTime();
    matrixMultKernel <<< dimGrid, dimBlock >>> (dev_m1, dev_m2, dev_res, N);
    stopKernelTime();

    startKernelTime();
	hipMemcpy(result, dev_res, N*N * sizeof(float), hipMemcpyDeviceToHost);
    checkCUDAError("Error using hipMemcpyHostToDevice");
    stopKernelTime();

    hipFree(dev_m1);
    hipFree(dev_m2);
	hipFree(dev_res);
}

int main (int argc, char** argv) {

	if(argc < 2)
		return 1;
	
	int N = atoi(argv[1]);
	float *m1, *m2, *result;

	fillMatrices (&m1, &m2, &result, N);

	matrixMultGPU (m1, m2, result, N);

	cout << validate(result, N) << endl;

	return 0;
}